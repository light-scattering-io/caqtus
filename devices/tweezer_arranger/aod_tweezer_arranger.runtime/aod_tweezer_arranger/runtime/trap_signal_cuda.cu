
#include <hip/hip_runtime.h>


#define PI 3.141592653589793
#define TAU 6.283185307179586
#define INV_PI 0.3183098861837907

#define MAX_NUMBER_TONES 100

__constant__ float frequencies[MAX_NUMBER_TONES];
__constant__ float amplitudes[MAX_NUMBER_TONES];
__constant__ float phases[MAX_NUMBER_TONES];

extern "C" __global__
void compute_static_traps_signal(short *output, unsigned int number_samples, unsigned int number_tones, float time_step)
{
 unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
 float result = 0.0;
 float x = time_step * TAU * tid;
 if (tid < number_samples){
   for(unsigned int i=0; i < number_tones; i++){
        float phase = x * frequencies[i] + phases[i];
        result +=  amplitudes[i] * __sinf(phase);
   }
   output[tid] = short(result * 32767.999);
 }
}

__constant__ float initial_frequencies[MAX_NUMBER_TONES];
__constant__ float final_frequencies[MAX_NUMBER_TONES];
__constant__ float initial_amplitudes[MAX_NUMBER_TONES];
__constant__ float final_amplitudes[MAX_NUMBER_TONES];
__constant__ float initial_phases[MAX_NUMBER_TONES];
__constant__ float final_phases[MAX_NUMBER_TONES];

// Must be equal to -1 at s=0 and +1 at s=1
__device__ float amplitude_ramp(float s)
{
    //return -1.0 + 2.0 * s;
    return -__cosf(PI * s);
} 

// Must be equal to -1 at s=0 and +1 at s=1
__device__ float frequency_ramp(float s)
{
    //return -1.0 + 2.0 * s;
    return -__cosf(PI * s);
}

__device__ float phase_ramp_sin(float s)
{
    return -__sinf(PI * s) * INV_PI;
}

__device__ float reach_constant_velocity_adiabatically(float s)
{
    float u = min(0.5, s);
    float v = min(0.25, s);
    
    return -2.0/3.0*powf(s, 4) + (8.0/3.0)*powf(s, 3) - 2*powf(s, 2) - 1.0/3.0*s + (4.0/3.0)*powf(min(1.0/2.0, s), 4) - 8.0/3.0*powf(min(1.0/2.0, s), 3) + 2*powf(min(1.0/2.0, s), 2) - 2.0/3.0*min(1.0/2.0, s);
}

__device__ float phase_ramp_minimal_jolt(float s)
{
    return ((s < 1.0/4.0) ? (
    (8.0/3.0)*powf(s, 4) - s
    )
    : ((s < 3.0/4.0) ? (
     -8.0/3.0*powf(s, 4) + (16.0/3.0)*powf(s, 3) - 2*powf(s, 2) - 2.0/3.0*s - 1.0/48.0
    )
    : (
    (8.0/3.0)*powf(s, 4) - 32.0/3.0*powf(s, 3) + 16*powf(s, 2) - 29.0/3.0*s + 5.0/3.0
    )));
}

// Must be the integral of frequency_ramp over s from 0 to s
__device__ float phase_ramp(float s, unsigned int move_type)
{
    if (move_type == 0)
        return phase_ramp_sin(s);   
    else if (move_type == 1)
        return phase_ramp_minimal_jolt(s);
    else if (move_type == 2)
        return reach_constant_velocity_adiabatically(s);
    else
        assert(0);
}



extern "C" __global__
void compute_moving_traps_signal(short *output, unsigned int number_samples, unsigned int number_tones, float time_step, unsigned int previous_step_stop, unsigned int next_step_start, unsigned int move_type)
{
 unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
 float s = float(tid) / float(number_samples);
 float result = 0.0;
 float T = time_step * number_samples;
 if (tid < number_samples){
   for(unsigned int i=0; i < number_tones; i++){
        float mean_frequency = 0.5 * (initial_frequencies[i] + final_frequencies[i]);
        float frequency_range = 0.5 * (final_frequencies[i] - initial_frequencies[i]);
        float initial_phase = initial_phases[i] + 2 * PI * previous_step_stop * time_step * initial_frequencies[i];
        float target_phases = final_phases[i] + 2 * PI * next_step_start * time_step * final_frequencies[i];
        float phase_mismatch = target_phases - initial_phase - (2 * PI * T) * mean_frequency;
        float s0=0.0;
        if(frequency_range == 0.0){
            s0 = 1.0;
        }
        else {
            float phase_remainder = fmodf(phase_mismatch, 2 * PI);
            s0 = (1.0 - phase_remainder / (2 * PI * T * frequency_range)) / (1-phase_ramp(1.0, move_type));
        }   
        float phase = 0.0;
        if(s < s0){
            phase = initial_phase +  2 * PI * T * (s * mean_frequency + frequency_range * s0 * phase_ramp(s / s0, move_type));
        }
        else{
            phase = initial_phase +  2 * PI * T * (s * mean_frequency + frequency_range * (s-s0 + s0 * phase_ramp(1.0, move_type)));
        }
        
        float mean_amplitude = 0.5 * (initial_amplitudes[i] + final_amplitudes[i]);
        float amplitude_range = 0.5 * (final_amplitudes[i] - initial_amplitudes[i]);
        float amplitude = mean_amplitude + amplitude_range * amplitude_ramp(s);
        result +=  amplitude * __sinf(phase);
   }
   output[tid] = short(result * 32767.999);
 }
}