#include "hip/hip_runtime.h"
#define PI 3.141592653589793
#define TAU 6.283185307179586
#define INV_PI 0.3183098861837907
#define MAX_NUMBER_TONES 400

__constant__ float frequencies[MAX_NUMBER_TONES];
__constant__ float amplitudes[MAX_NUMBER_TONES];
__constant__ float phases[MAX_NUMBER_TONES];

extern "C" __global__
void
compute_static_traps_signal(short *output, unsigned int number_samples, unsigned int number_tones, float time_step) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float result = 0.0;
    float x = time_step * TAU * tid;
    if (tid < number_samples) {
        for (unsigned int i = 0; i < number_tones; i++) {
            float phase = x * frequencies[i] + phases[i];
            result += amplitudes[i] * __sinf(phase);
        }
        output[tid] = short(result * 32767.999);
    }
}

__constant__ float initial_frequencies[MAX_NUMBER_TONES];
__constant__ float final_frequencies[MAX_NUMBER_TONES];
__constant__ float initial_amplitudes[{max_number_tones}];
__constant__ float final_amplitudes[{max_number_tones}];
__constant__ float initial_phases[{max_number_tones}];
__constant__ float final_phases[{max_number_tones}];


// Must be equal to -1 at s=0 and +1 at s=1
__device__ float amplitude_ramp(float s) {
    return -__cosf(PI * s);
}

// Must be equal to -1 at s=0 and +1 at s=1
__device__ float frequency_ramp(float s) {
    return -__cosf(PI * s);
}

__device__ float phase_ramp_sin(float s) {
    return -__sinf(PI * s) * INV_PI;
}

__device__ float reach_constant_velocity_adiabatically(float s) {
    return -2.0 / 3.0 * powf(s, 4) + (8.0 / 3.0) * powf(s, 3) - 2 * powf(s, 2) - 1.0 / 3.0 * s +
           (4.0 / 3.0) * powf(min(1.0 / 2.0, s), 4) - 8.0 / 3.0 * powf(min(1.0 / 2.0, s), 3) +
           2 * powf(min(1.0 / 2.0, s), 2) - 2.0 / 3.0 * min(1.0 / 2.0, s);
}


__device__ float phase_ramp_minimal_jolt(float s) {
    return ((s < 1.0 / 4.0) ? (
            (8.0 / 3.0) * powf(s, 4) - s
    )
                            : ((s < 3.0 / 4.0) ? (
                    -8.0 / 3.0 * powf(s, 4) + (16.0 / 3.0) * powf(s, 3) - 2 * powf(s, 2) - 2.0 / 3.0 * s -
                    1.0 / 48.0
            )
                                               : (
                                       (8.0 / 3.0) * powf(s, 4) - 32.0 / 3.0 * powf(s, 3) + 16 * powf(s, 2) -
                                       29.0 / 3.0 * s + 5.0 / 3.0
                               )));
}

// Must be the integral of frequency_ramp over s from 0 to s
__device__ float phase_ramp(float s) {
    return reach_constant_velocity_adiabatically(s);
    //return phase_ramp_sin(s);
}


extern "C" __global__
void compute_moving_traps_signal(short *output, unsigned int number_samples, unsigned int number_tones, float time_step,
                                 unsigned int previous_step_stop, unsigned int next_step_start) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float s = float(tid) / float(number_samples);
    float result = 0.0;
    float T = time_step * number_samples;
    if (tid < number_samples) {
        for (unsigned int i = 0; i < number_tones; i++) {

            float mean_frequency = 0.5 * (initial_frequencies[i] + final_frequencies[i]);
            float frequency_range = 0.5 * (final_frequencies[i] - initial_frequencies[i]);
            float initial_phase =
                    initial_phases[i] + 2 * PI * previous_step_stop * time_stóep * initial_frequencies[i];
            float target_phases =
                    final_phases[i] + 2 * PI * next_step_start * time_step * final_frequencies[i];
            float phase_mismatch = target_phases - initial_phase - (2 * PI * T) * mean_frequency;
            float s0 = 0.0;
            if (frequency_range == 0.0)
                s0 = 1.0;
            else {
                float phase_remainder = fmodf(phase_mismatch, 2 * PI);
                s0 = (1.0 - phase_remainder / (2 * PI * T * frequency_range)) / (1 - phase_ramp(1.0));
            }
            float phase = 0.0;
            if (s < s0) {
                phase = initial_phase +
                        2 * PI * T * (s * mean_frequency + frequency_range * s0 * phase_ramp(s / s0));
            } else {

                phase = initial_phase + 2 * PI * T * (s * mean_frequency + frequency_range * (s - s0 +
                                                                                              s0 *
                                                                                              phase_ramp(
                                                                                                      1.0)));

            }
            float mean_amplitude = 0.5 * (initial_amplitudes[i] + final_amplitudes[i]);
            float amplitude_range = 0.5 * (final_amplitudes[i] - initial_amplitudes[i]);
            float amplitude = mean_amplitude + amplitude_range * amplitude_ramp(s);
            result += amplitude * __sinf(phase);
        }
        output[tid] = short(result * 32767.999);
    }
}